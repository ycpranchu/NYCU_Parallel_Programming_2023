#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(int* result, int resX, int resY, float lowerX, float lowerY, float dx, float dy, int maxIterations)
{
	int thisX = blockIdx.x * blockDim.x + threadIdx.x;
	int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    if (thisX >= resX) return;
    if (thisY >= resY) return;

    float c_re = lowerX + thisX * dx;
	float c_im = lowerY + thisY * dy;
	float z_re = c_re;
	float z_im = c_im;
    int i;

    #pragma unroll
    for (i = 0; i < maxIterations; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f) break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    int index = thisX + thisY * resX;
    result[index] = i;
}

void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
	float dx = (upperX - lowerX) / resX;
    float dy = (upperY - lowerY) / resY;
    int size = resX * resY * sizeof(int);

    int *hostMemory = (int *)malloc(size);
    int *result;
    hipMalloc(&result, size);

    dim3 blockSize(32, 32);
    dim3 numBlocks(resX / blockSize.x + 1, resY / blockSize.y + 1);

    mandelKernel<<<numBlocks, blockSize>>> (result, resX, resY, lowerX, lowerY, dx, dy, maxIterations);

    hipMemcpy(hostMemory, result, size, hipMemcpyDeviceToHost);
    memcpy(img, hostMemory, size);
    
    hipFree(result);
    free(hostMemory);
}
