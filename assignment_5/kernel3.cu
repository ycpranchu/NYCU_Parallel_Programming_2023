#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
using byte = unsigned char;

__global__ void mandelKernel(int* result, int resX, int resY, float lowerX, float lowerY, float dx, float dy, int maxIterations, int pitch, int group_size)
{
	int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * group_size;
	int thisY = (blockIdx.y * blockDim.y + threadIdx.y) * group_size;

    if (thisX >= resX) return;
    if (thisY >= resY) return;
    
    for (int offset_y = 0; offset_y < group_size; offset_y++)
    {
        for (int offset_x = 0; offset_x < group_size; offset_x++)
        { 
            float c_re = lowerX + (thisX + offset_x) * dx;
            float c_im = lowerY + (thisY + offset_y) * dy;
            float z_re = c_re;
            float z_im = c_im;
            int i;

            #pragma unroll
            for (i = 0; i < maxIterations; ++i)
            {
                if (z_re * z_re + z_im * z_im > 4.f) break;

                float new_re = z_re * z_re - z_im * z_im;
                float new_im = 2.f * z_re * z_im;
                z_re = c_re + new_re;
                z_im = c_im + new_im;
            }

            int* row = (int*)((byte*)result + (thisY + offset_y) * pitch); // char size = 8-bit
            row[(thisX + offset_x)] = i;
        }
    }
}

void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
	float dx = (upperX - lowerX) / resX;
    float dy = (upperY - lowerY) / resY;
    int size = resX * resY * sizeof(int);
    int group_size = 10;

    int *hostMemory, *result;
    size_t pitch = 0;

    hipHostAlloc(&hostMemory, size, hipHostMallocMapped);
    hipMallocPitch(&result, &pitch, sizeof(int) * resX, resY);

    dim3 blockSize(32, 32);
    dim3 numBlocks(resX / blockSize.x / group_size + 1, resY / blockSize.y / group_size + 1);
    mandelKernel<<<numBlocks, blockSize>>> (result, resX, resY, lowerX, lowerY, dx, dy, maxIterations, pitch, group_size);

    hipMemcpy2D(hostMemory, sizeof(int) * resX, result, pitch, sizeof(int) * resX, resY, hipMemcpyDeviceToHost);
    memcpy(img, hostMemory, size);
    
    hipFree(result);
    hipHostFree(hostMemory);
}
