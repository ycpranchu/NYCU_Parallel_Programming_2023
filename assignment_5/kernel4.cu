#pragma optimize(3, "Ofast", "inline")
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

template <int maxIterations>

__global__ void mandelKernel(int* result, int resX, int resY, float lowerX, float lowerY, float dx, float dy)
{
	int thisX = blockIdx.x * blockDim.x + threadIdx.x;
	int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    if (thisX >= resX || thisY >= resY) return;

    float c_re = lowerX + thisX * dx;
	float c_im = lowerY + thisY * dy;
	float z_re = c_re, z_im = c_im;
    int i = 0;

    #pragma unroll
    for (; i < maxIterations; ++i) {
        float a = z_re * z_re;
        float b = z_im * z_im;

        if (a + b > 4.f) break;

        float new_re = a - b;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    result[thisX + thisY * resX] = i;
}

__global__ void mandelKernelDynamic(int* result, int resX, int resY, float lowerX, float lowerY, float dx, float dy, int maxIterations)
{
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
	int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    if (thisX >= resX || thisY >= resY) return;

    float c_re = lowerX + thisX * dx;
	float c_im = lowerY + thisY * dy;
	float z_re = c_re, z_im = c_im;
    int i = 0;

    #pragma unroll
    for (; i < maxIterations; ++i) {
        float a = z_re * z_re;
        float b = z_im * z_im;

        if (a + b > 4.f) break;

        float new_re = a - b;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    result[thisX + thisY * resX] = i;
}

void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
	float dx = (upperX - lowerX) / resX;
    float dy = (upperY - lowerY) / resY;
    int size = resX * resY * sizeof(int);

    int *result;
    hipMalloc(&result, size);

    dim3 blockSize(8, 8);
    dim3 numBlocks(resX / blockSize.x + 1, resY / blockSize.y + 1);

    switch (maxIterations) {
        case 256:
            mandelKernel<256><<<numBlocks, blockSize>>>(result, resX, resY, lowerX, lowerY, dx, dy);
            break;
        case 1000:
            mandelKernel<1000><<<numBlocks, blockSize>>>(result, resX, resY, lowerX, lowerY, dx, dy);
            break;
        case 10000:
            mandelKernel<10000><<<numBlocks, blockSize>>>(result, resX, resY, lowerX, lowerY, dx, dy);
            break;
        case 100000:
            mandelKernel<100000><<<numBlocks, blockSize>>>(result, resX, resY, lowerX, lowerY, dx, dy);
            break;
        default:
            mandelKernelDynamic<<<numBlocks, blockSize>>>(result, resX, resY, lowerX, lowerY, dx, dy, maxIterations);
            break;
    }

    hipMemcpy(img, result, size, hipMemcpyDeviceToHost);
    hipFree(result);
}
